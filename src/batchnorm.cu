#include "hip/hip_runtime.h"
#include <stdio.h>
#include "xcuda.h"
#include "xallocs.h"
#include "utils.h"
#include <math.h>
#include "network.h"


#ifdef __INTELLISENSE__
#define KARGS(...)
#define __syncthreads()
#else
#define KARGS(...) <<< __VA_ARGS__ >>>
#endif


__global__ void forward_batchnorm_kernel(float* Z, int batch_size, int n_filters, int out_wh, float* mean) {
	// Steps:
	// Calculate means of each filter.
	// Calculate variances of each filter.
	// Calculate new rolling means and rolling variances.
	// Calculate z-norm then apply scale (gamma) and shift (beta).


	// calculate means
	

}

__global__ void batchnorm_kernel(float* gammas, float* betas, float* means, float* variances, float* rolling_means, float* rolling_variances, float* Z, float* Z_norm, float* act_inputs, int spatial, int batch_size, int out_n) {

	__shared__ extern float shared[];
	
	int tid = threadIdx.x;
	int filter = blockIdx.x;
	int block_size = blockDim.x;

	int fst = filter * spatial + tid;

	// Calculate means
	shared[tid] = 0.0F;
	// Copy/Add data to shared memory
	for (int b = 0; b < batch_size; b++) {
		int offset = b * out_n + fst;
		for (int s = 0; s < spatial; s += block_size) {
			if (s + tid < spatial) shared[tid] += Z[offset + s];
		}
	}
	__syncthreads();

	// Parallel reduction sum
	for (int stride = block_size / 2; stride > 0; stride >>= 1) {
		if (tid < stride) {
			shared[tid] += shared[tid + stride];
		}
		__syncthreads();
	}

	float mean = shared[0] / (float)(spatial * batch_size);
	if (tid == 0) {
		means[filter] = mean;
		rolling_means[filter] = (mean * 0.01F) + (rolling_means[filter] * 0.99F);
	}

	// Calculate variances
	shared[tid] = 0.0F;
	for (int b = 0; b < batch_size; b++) {
		int offset = b * out_n + fst;
		for (int s = 0; s < spatial; s += block_size) {
			if (s + tid < spatial) shared[tid] += powf(Z[offset + s] - mean, 2.0F);
		}
	}
	__syncthreads();

	// Parallel reduction sum
	for (int stride = block_size / 2; stride > 0; stride >>= 1) {
		if (tid < stride) {
			shared[tid] += shared[tid + stride];
		}
		__syncthreads();
	}

	float variance = shared[0] / (float)(spatial * batch_size);
	if (tid == 0) {
		variances[filter] = variance;
		rolling_variances[filter] = (variance * 0.01F) + (rolling_variances[filter] * 0.99F);
	}

	// Normalize values
	float gamma = gammas[filter];
	float beta = betas[filter];
	float sddev = sqrtf(variance + 0.00001F);
	for (int b = 0; b < batch_size; b++) {
		int offset = b * out_n + fst;
		for (int s = 0; s < spatial; s += block_size) {
			int z = offset + s;
			if (s + tid < spatial) {
				float znorm = (Z[z] - mean) / sddev;
				Z_norm[z] = znorm;
				act_inputs[z] = znorm * gamma + beta;
			}
		}
	}
}

void test_forward_batchnorm_gpu(layer* l, size_t batch_size) {
	float* Z = l->Z;
	float* Z_norm = l->Z_norm;
	float* act_inputs = l->act_inputs;
	float* means = l->means;
	float* variances = l->variances;
	float* gammas = l->gammas;
	float* betas = l->biases;
	float* rolling_means = l->rolling_means;
	float* rolling_variances = l->rolling_variances;
	size_t F = l->n_filters;
	size_t S = l->out_w * l->out_h;
	size_t B = batch_size;
	size_t out_n = l->out_n;

	float* d_Z = 0;
	CHECK_CUDA(hipMalloc(&d_Z, F * S * B * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_Z, Z, F * S * B * sizeof(float), hipMemcpyHostToDevice));

	int grid_size = (int)F;
	int block_size = 512;
	int shared_mem_size = (int)(S * B) * sizeof(float);
	batchnorm_kernel KARGS(grid_size, block_size, shared_mem_size) (gammas, betas, means, variances, rolling_means, rolling_variances, Z, Z_norm, act_inputs, S, batch_size, out_n);
}