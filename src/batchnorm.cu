#include "hip/hip_runtime.h"
#include <stdio.h>
#include "xcuda.h"
#include "xallocs.h"
#include "utils.h"
#include <math.h>
#include "network.h"
#include "batchnorm.h"


#ifdef __INTELLISENSE__
#define KARGS(...)
#define __syncthreads()
#define __shfl_down_sync(...) ( __VA_ARGS__ )
#else
#define KARGS(...) <<< __VA_ARGS__ >>>
#endif


__global__ void forward_batchnorm_kernel_no_shuffle(
	const float* __restrict__ gammas, const float* __restrict__ betas,
	float* means, float* variances,
	float* rolling_means, float* rolling_variances,
	float* __restrict__ Z, float* Z_norm, float* act_inputs,
	int spatial, int n, int batch_size) {

	__shared__ float shared[BLOCKSIZE];
	
	int tid = threadIdx.x;
	int filter = blockIdx.x;
	int block_size = blockDim.x;

	int fs = filter * spatial;

	// Calculate means
	shared[tid] = 0.0F;
	// Copy/Add data to shared memory
	for (int b = 0; b < batch_size; b++) {
		int offset = b * n + fs;
		for (int s = tid; s < spatial; s += block_size) {
			shared[tid] += Z[offset + s];
		}
	}
	__syncthreads();

	// Parallel reduction sum
	for (int stride = block_size >> 1; stride > 0; stride >>= 1) {
		if (tid < stride) {
			shared[tid] += shared[tid + stride];
		}
		__syncthreads();
	}

	float mean = shared[0] / (float)(spatial * batch_size);
	if (tid == 0) {
		means[filter] = mean;
		rolling_means[filter] = (mean * 0.01F) + (rolling_means[filter] * 0.99F);
	}

	// Calculate variances
	shared[tid] = 0.0F;
	for (int b = 0; b < batch_size; b++) {
		int offset = b * n + fs;
		for (int s = tid; s < spatial; s += block_size) {
			float dev = Z[offset + s] - mean;
			shared[tid] += dev * dev;
		}
	}
	__syncthreads();

	// Parallel reduction sum
	for (int stride = block_size >> 1; stride > 0; stride >>= 1) {
		if (tid < stride) {
			shared[tid] += shared[tid + stride];
		}
		__syncthreads();
	}

	float variance = shared[0] / (float)(spatial * batch_size);
	if (tid == 0) {
		variances[filter] = variance;
		rolling_variances[filter] = (variance * 0.01F) + (rolling_variances[filter] * 0.99F);
	}

	// Normalize values
	float gamma = gammas[filter];
	float beta = betas[filter];
	float sddev = sqrtf(variance + 0.00001F);
	for (int b = 0; b < batch_size; b++) {
		int offset = b * n + fs;
		for (int s = tid; s < spatial; s += block_size) {
			int z = offset + s;
			float znorm = (Z[z] - mean) / sddev;
			Z_norm[z] = znorm;
			act_inputs[z] = znorm * gamma + beta;
		}
	}
}

__global__ void forward_batchnorm_kernel(
	const float* __restrict__ gammas, const float* __restrict__ betas,
	float* means, float* variances,
	float* rolling_means, float* rolling_variances,
	float* __restrict__ Z, float* Z_norm, float* act_inputs,
	int spatial, int n, int batch_size) {

	__shared__ float warp_sums[BLOCKSIZE >> 5]; // # of warps per block
	int tid = threadIdx.x;
	int filter = blockIdx.x;
	
	int lane = threadIdx.x & 31;
	int warp_id = threadIdx.x >> 5;

	int fs = filter * spatial;

	float thread_sum = 0.0F;

	// MEANS
	// Each thread computes a partial sum
	for (int b = 0; b < batch_size; b++) {
		int offset = b * n + fs;
		for (int s = tid; s < spatial; s += BLOCKSIZE) {
			if (s < spatial) thread_sum += Z[offset + s];
		}
	}

	// In-warp reduction using shuffle
	for (int offset = 16; offset > 0; offset >>= 1) {
		thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
	}
	// Warp leaders write to shared memory
	if (lane == 0) warp_sums[warp_id] = thread_sum;
	__syncthreads();

	// First warp reduces warp_sums[] to get block total
	float block_sum = 0.0F;
	if (warp_id == 0) {
		block_sum = (tid < (BLOCKSIZE >> 5)) ? warp_sums[tid] : 0.0F;
		for (int offset = 16; offset > 0; offset >>= 1) {
			block_sum += __shfl_down_sync(0xffffffff, block_sum, offset);
		}
		if (tid == 0) {
			float mean = block_sum / (float)(spatial * batch_size);
			warp_sums[0] = mean;
			means[filter] = mean;
			rolling_means[filter] = mean * 0.01F + rolling_means[filter] * 0.99F;
		}
	}
	__syncthreads();
	float mean = warp_sums[0];

	// VARIANCES
	thread_sum = 0.0F;
	for (int b = 0; b < batch_size; b++) {
		int offset = b * n + fs;
		for (int s = tid; s < spatial; s += BLOCKSIZE) {
			float dev = Z[offset + s] - mean;
			thread_sum += dev * dev;
		}
	}
	
	// In-warp reduction using shuffle
	for (int offset = 16; offset > 0; offset >>= 1) {
		thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
	}
	if (lane == 0) warp_sums[warp_id] = thread_sum;
	__syncthreads();

	// First warp reduces warp_sums[] to get block total
	block_sum = 0.0F;
	if (warp_id == 0) {
		block_sum = (tid < (BLOCKSIZE >> 5)) ? warp_sums[tid] : 0.0f;
		for (int offset = 16; offset > 0; offset >>= 1) {
			block_sum += __shfl_down_sync(0xffffffff, block_sum, offset);
		}
		if (tid == 0) {
			float variance = block_sum / (float)(spatial * batch_size);
			warp_sums[0] = variance;
			variances[filter] = variance;
			rolling_variances[filter] = (variance * 0.01F) + (rolling_variances[filter] * 0.99F);
		}
	}
	__syncthreads();
	float variance = warp_sums[0];

	// NORMALIZE AND AFFINE
	float gamma = gammas[filter];
	float beta = betas[filter];
	float sddev = sqrtf(variance + 0.00001F);
	for (int b = 0; b < batch_size; b++) {
		int offset = b * n + fs;
		for (int s = tid; s < spatial; s += BLOCKSIZE) {
			int z = offset + s;
			float znorm = (Z[z] - mean) / sddev;
			Z_norm[z] = znorm;
			act_inputs[z] = znorm * gamma + beta;
		}
	}
}

void forward_batchnorm_gpu(float* gammas, float* betas,
	float* means, float* variances,
	float* rolling_means, float* rolling_variances,
	float* Z, float* Z_norm, float* act_inputs,
	int spatial, int n_filters, int batch_size) {

	int n = spatial * n_filters;
	forward_batchnorm_kernel KARGS(n_filters, BLOCKSIZE) (gammas, betas, means, variances, rolling_means, rolling_variances, Z, Z_norm, act_inputs, spatial, batch_size, n);
	CHECK_CUDA(hipPeekAtLastError());
}

void test_forward_batchnorm_gpu(void) {
	int batch_size = 8;
	int w = 320;
	int h = 320;
	int spatial = w * h;
	int n_filters = 64;
	int out_n = spatial * n_filters;
	float* Z = (float*)xcalloc(out_n * batch_size, sizeof(float));
	float* Z_norm = (float*)xcalloc(out_n * batch_size, sizeof(float));
	float* act_inputs = (float*)xcalloc(out_n * batch_size, sizeof(float));
	float* means = (float*)xcalloc(n_filters, sizeof(float));
	float* variances = (float*)xcalloc(n_filters, sizeof(float));
	float* gammas = (float*)xcalloc(n_filters, sizeof(float));
	float* betas = (float*)xcalloc(n_filters, sizeof(float));
	float* rolling_means = (float*)xcalloc(n_filters, sizeof(float));
	float* rolling_variances = (float*)xcalloc(n_filters, sizeof(float));

	fill_array_rand_float(Z, out_n * batch_size, 0.0F, 1.0F);
	float* d_Z = 0;
	CHECK_CUDA(hipMalloc(&d_Z, out_n * batch_size * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_Z, Z, out_n * batch_size * sizeof(float), hipMemcpyHostToDevice));

	float* d_act_inputs = 0;
	CHECK_CUDA(hipMalloc(&d_act_inputs, out_n * batch_size * sizeof(float)));

	float* d_Z_norm = 0;
	CHECK_CUDA(hipMalloc(&d_Z_norm, out_n * batch_size * sizeof(float)));

	fill_array_rand_float(means, n_filters, 0.0F, 1.0F);
	float* d_means = 0;
	CHECK_CUDA(hipMalloc(&d_means, n_filters * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_means, means, n_filters * sizeof(float), hipMemcpyHostToDevice));

	fill_array_rand_float(means, n_filters, 0.0F, 1.0F);
	float* d_variances = 0;
	CHECK_CUDA(hipMalloc(&d_variances, n_filters * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_variances, variances, n_filters * sizeof(float), hipMemcpyHostToDevice));

	fill_array_rand_float(gammas, n_filters, 0.0F, 1.0F);
	float* d_gammas = 0;
	CHECK_CUDA(hipMalloc(&d_gammas, n_filters * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_gammas, gammas, n_filters * sizeof(float), hipMemcpyHostToDevice));

	fill_array_rand_float(betas, n_filters, 0.0F, 1.0F);
	float* d_betas = 0;
	CHECK_CUDA(hipMalloc(&d_betas, n_filters * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_betas, betas, n_filters * sizeof(float), hipMemcpyHostToDevice));

	fill_array_rand_float(rolling_means, n_filters, 0.0F, 1.0F);
	float* d_rolling_means = 0;
	CHECK_CUDA(hipMalloc(&d_rolling_means, n_filters * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_rolling_means, rolling_means, n_filters * sizeof(float), hipMemcpyHostToDevice));

	fill_array_rand_float(rolling_variances, n_filters, 0.0F, 1.0F);
	float* d_rolling_variances = 0;
	CHECK_CUDA(hipMalloc(&d_rolling_variances, n_filters * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_rolling_variances, rolling_variances, n_filters * sizeof(float), hipMemcpyHostToDevice));

	int grid_size = n_filters;
	int block_size = BLOCKSIZE;
	int select = 2;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	if (select == 1) {
		forward_batchnorm_kernel_no_shuffle KARGS(grid_size, block_size) (d_gammas, d_betas, d_means, d_variances, d_rolling_means, d_rolling_variances, d_Z, d_Z_norm, d_act_inputs, spatial, batch_size, out_n);
	}
	else {
		forward_batchnorm_kernel KARGS(grid_size, block_size) (d_gammas, d_betas, d_means, d_variances, d_rolling_means, d_rolling_variances, d_Z, d_Z_norm, d_act_inputs, spatial, batch_size, out_n);
	}

	CHECK_CUDA(hipGetLastError());

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("batchnorm kernel execution time: %f ms\n", milliseconds);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	CHECK_CUDA(hipGetLastError());
	CHECK_CUDA(hipDeviceSynchronize());

	CHECK_CUDA(hipMemcpy(act_inputs, d_act_inputs, out_n * batch_size * sizeof(float), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_Z));
	CHECK_CUDA(hipMemcpy(Z_norm, d_Z_norm, out_n * batch_size * sizeof(float), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_Z_norm));
	CHECK_CUDA(hipMemcpy(means, d_means, n_filters * sizeof(float), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_means));
	CHECK_CUDA(hipMemcpy(variances, d_variances, n_filters * sizeof(float), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_variances));
	CHECK_CUDA(hipMemcpy(gammas, d_gammas, n_filters * sizeof(float), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_gammas));
	CHECK_CUDA(hipMemcpy(betas, d_betas, n_filters * sizeof(float), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_betas));
	CHECK_CUDA(hipMemcpy(rolling_means, d_rolling_means, n_filters * sizeof(float), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_rolling_means));
	CHECK_CUDA(hipMemcpy(rolling_variances, d_rolling_variances, n_filters * sizeof(float), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_rolling_variances));

	layer l = { 0 };
	l.out_w = (size_t)w;
	l.out_h = (size_t)h;
	l.n_filters = (size_t)n_filters;
	l.out_n = (size_t)out_n;
	l.Z = Z;
	l.Z_norm = Z_norm;
	l.act_inputs = (float*)xcalloc(out_n * batch_size, sizeof(float));
	l.means = means;
	l.variances = variances;
	l.gammas = gammas;
	l.biases = betas;
	l.rolling_means = rolling_means;
	l.rolling_variances = rolling_variances;
	forward_batchnorm(&l, (size_t)batch_size);

	float epsilon = 1e-2f;
	printf("Verifiying......\n");
	for (size_t i = 0; i < out_n * batch_size; i++) {
		//printf("%f : %f\n", l.act_inputs[i], act_inputs[i]);
		if (fabs(l.act_inputs[i] - act_inputs[i]) > epsilon || isnan(l.act_inputs[i]) || isnan(act_inputs[i])) {
			printf("Verification Failed: i = %zu, (cpu)%f != (gpu)%f\n", i, l.act_inputs[i], act_inputs[i]);
			wait_for_key_then_exit();
		}
	}
	printf("Verifiction Success!!!\n");
}

__global__ void backward_batchnorm_kernel(
	float* grads,
	float* Z, float* Z_norm,
	float* means, float* variances,
	float* gammas, float* gamma_grads,
	int spatial, int n, int batch_size)
	{

	__shared__ float warp_sums[BLOCKSIZE >> 5];
	__shared__ float gamma;
	__shared__ float mean;
	__shared__ float variance;
	__shared__ float mean_grad;
	__shared__ float variance_grad;

	int tid = threadIdx.x;
	int filter = blockIdx.x;

	int lane = threadIdx.x & 31;
	int warp_id = threadIdx.x >> 5;

	int fs = filter * spatial;

	// --- GAMMA_GRADS ---
	float thread_sum = 0.0F;
	for (int b = 0; b < batch_size; b++) {
		int offset = b * n + fs;
		for (int s = tid; s < spatial; s += BLOCKSIZE) {
			int i = offset + s;
			thread_sum += grads[i] * Z_norm[i];
			Z_norm[i] = 0.0F;
		}
	}

	for (int offset = 16; offset > 0; offset >>= 1) {
		thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
	}

	if (lane == 0) warp_sums[warp_id] = thread_sum;
	if (tid == 0) {
		gamma = gammas[filter];
		mean = means[filter];
		variance = variances[filter];
	}
	__syncthreads();

	if (warp_id == 0) {
		float block_sum = (tid < BLOCKSIZE >> 5) ? warp_sums[tid] : 0.0F;
		for (int offset = 16; offset > 0; offset >>= 1) {
			block_sum += __shfl_down_sync(0xffffffff, block_sum, offset);
		}
		if (tid == 0) {
			gamma_grads[filter] = block_sum;
		}
	}
	
	// --- MEAN_GRADS ---
	thread_sum = 0.0F;
	for (int b = 0; b < batch_size; b++) {
		int offset = b * n + fs;
		for (int s = tid; s < spatial; s += BLOCKSIZE) {
			float grad = grads[offset + s] * gamma;
			grads[offset + s] = grad;
			thread_sum += grad;
		}
	}

	for (int offset = 16; offset > 0; offset >>= 1) {
		thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
	}

	if (lane == 0) warp_sums[warp_id] = thread_sum;
	__syncthreads();

	if (warp_id == 0) {
		float block_sum = (tid < BLOCKSIZE >> 5) ? warp_sums[tid] : 0.0F;
		for (int offset = 16; offset > 0; offset >>= 1) {
			block_sum += __shfl_down_sync(0xffffffff, block_sum, offset);
		}
		if (tid == 0) {
			mean_grad = block_sum * (-1.0F / sqrtf(variance + 0.00001F));
		}
	}
	__syncthreads();

	// --- VARIANCE_GRADS ---
	thread_sum = 0.0F;
	for (int b = 0; b < batch_size; b++) {
		int offset = b * n + fs;
		for (int s = tid; s < spatial; s += BLOCKSIZE) {
			int i = offset + s;
			thread_sum += grads[i] * (Z[i] - mean);
		}
	}

	for (int offset = 16; offset > 0; offset >>= 1) {
		thread_sum += __shfl_down_sync(0xffffffff, thread_sum, offset);
	}

	if (lane == 0) warp_sums[warp_id] = thread_sum;
	__syncthreads();

	if (warp_id == 0) {
		float block_sum = (tid < BLOCKSIZE >> 5) ? warp_sums[tid] : 0.0F;
		for (int offset = 16; offset > 0; offset >>= 1) {
			block_sum += __shfl_down_sync(0xffffffff, block_sum, offset);
		}
		if (tid == 0) {
			variance_grad = block_sum * -0.5F * powf(variance + 0.00001F, (float)(-3.0F / 2.0F));
		}
	}
	__syncthreads();

	// --- GRADS ---
	float sb = (float)(spatial * batch_size);
	for (int b = 0; b < batch_size; b++) {
		int offset = b * n + fs;
		for (int s = tid; s < spatial; s += BLOCKSIZE) {
			int i = offset + s;
			float grad = grads[i];
			grads[i] = grad * 1.0F / sqrtf(variance + 0.00001F) + variance_grad * 2.0F * (grad - mean) / sb + mean_grad / sb;
		}
	}
}

void backward_batchnorm_gpu(float* grads,
	float* Z, float* Z_norm,
	float* means, float* variances,
	float* gammas, float* gamma_grads,
	int spatial, int n_filters, int batch_size) {

	int n = spatial * n_filters;
	int grid_size = n_filters;
	backward_batchnorm_kernel KARGS(grid_size, BLOCKSIZE) (grads, Z, Z_norm, means, variances, gammas, gamma_grads, spatial, n, batch_size);
	CHECK_CUDA(hipPeekAtLastError());
}

void test_backward_batchnorm_gpu(void) {
	int batch_size = 8;
	int w = 320;
	int h = 320;
	int spatial = w * h;
	int n_filters = 64;
	int out_n = spatial * n_filters;
	float* Z = (float*)xcalloc(out_n * batch_size, sizeof(float));
	float* Z_norm = (float*)xcalloc(out_n * batch_size, sizeof(float));
	float* means = (float*)xcalloc(n_filters, sizeof(float));
	float* variances = (float*)xcalloc(n_filters, sizeof(float));
	float* gammas = (float*)xcalloc(n_filters, sizeof(float));
	float* gamma_grads = (float*)xcalloc(n_filters, sizeof(float));
	float* grads = (float*)xcalloc(out_n * batch_size, sizeof(float));
	
	fill_array_rand_float(Z, out_n * batch_size, 0.0F, 0.5F);
	float* d_Z = 0;
	CHECK_CUDA(hipMalloc(&d_Z, out_n * batch_size * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_Z, Z, out_n * batch_size * sizeof(float), hipMemcpyHostToDevice));

	fill_array_rand_float(Z_norm, out_n * batch_size, 0.0F, 0.1F);
	float* d_Z_norm = 0;
	CHECK_CUDA(hipMalloc(&d_Z_norm, out_n * batch_size * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_Z_norm, Z_norm, out_n * batch_size * sizeof(float), hipMemcpyHostToDevice));

	fill_array_rand_float(means, n_filters, 0.0F, 0.1F);
	float* d_means = 0;
	CHECK_CUDA(hipMalloc(&d_means, n_filters * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_means, means, n_filters * sizeof(float), hipMemcpyHostToDevice));

	fill_array_rand_float(variances, n_filters, 0.0F, 0.1F);
	for (int i = 0; i < n_filters; ++i) variances[i] = fabsf(variances[i]);
	float* d_variances = 0;
	CHECK_CUDA(hipMalloc(&d_variances, n_filters * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_variances, variances, n_filters * sizeof(float), hipMemcpyHostToDevice));

	fill_array_rand_float(gammas, n_filters, 0.0F, 0.5F);
	float* d_gammas = 0;
	CHECK_CUDA(hipMalloc(&d_gammas, n_filters * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_gammas, gammas, n_filters * sizeof(float), hipMemcpyHostToDevice));

	float* d_gamma_grads = 0;
	CHECK_CUDA(hipMalloc(&d_gamma_grads, n_filters * sizeof(float)));

	fill_array_rand_float(grads, out_n * batch_size, 0.0F, 0.01F);
	float* d_grads = 0;
	CHECK_CUDA(hipMalloc(&d_grads, out_n * batch_size * sizeof(float)));
	CHECK_CUDA(hipMemcpy(d_grads, grads, out_n * batch_size * sizeof(float), hipMemcpyHostToDevice));

	int grid_size = n_filters;
	int block_size = BLOCKSIZE;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	backward_batchnorm_kernel KARGS(grid_size, block_size) (d_grads, d_Z, d_Z_norm, d_means, d_variances, d_gammas, d_gamma_grads, spatial, out_n, batch_size);
	
	CHECK_CUDA(hipGetLastError());

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("batchnorm kernel execution time: %f ms\n", milliseconds);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	CHECK_CUDA(hipGetLastError());
	CHECK_CUDA(hipDeviceSynchronize());

	float* gpu_grads = (float*)xmalloc(out_n * batch_size * sizeof(float));
	CHECK_CUDA(hipMemcpy(gpu_grads, d_grads, out_n * batch_size * sizeof(float), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_grads));
	CHECK_CUDA(hipMemcpy(gamma_grads, d_gamma_grads, n_filters * sizeof(float), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_gamma_grads));

	CHECK_CUDA(hipFree(d_Z));
	CHECK_CUDA(hipFree(d_Z_norm));
	CHECK_CUDA(hipFree(d_means));
	CHECK_CUDA(hipFree(d_variances));
	CHECK_CUDA(hipFree(d_gammas));

	layer l = { 0 };

	l.out_w = (size_t)w;
	l.out_h = (size_t)h;
	l.n_filters = (size_t)n_filters;
	l.out_n = (size_t)out_n;
	l.Z = Z;
	l.Z_norm = Z_norm;
	l.means = means;
	l.variances = variances;
	l.gammas = gammas;
	l.grads = grads;
	l.gamma_grads = (float*)xmalloc(n_filters * sizeof(float));
	
	backward_batchnorm(&l, (size_t)batch_size);

	float epsilon = 2e-2f;
	printf("Verifiying grads......\n");
	for (size_t i = 0; i < out_n * batch_size; i++) {
		//printf("%f : %f\n", l.grads[i], gpu_grads[i]);
		if (fabsf(l.grads[i] - gpu_grads[i]) > epsilon || isnan(l.grads[i]) || isnan(gpu_grads[i])) {
			printf("Verification Failed: i = %zu, (cpu)%f != (gpu)%f\n", i, l.grads[i], gpu_grads[i]);
			wait_for_key_then_exit();
		}
	}
	printf("Grads verification success.\n");
	printf("Verifiying gamma grads......\n");
	for (size_t i = 0; i < n_filters; i++) {
		//printf("%f : %f\n", l.gamma_grads[i], gamma_grads[i]);
		if (fabsf(l.gamma_grads[i] - gamma_grads[i]) > epsilon || isnan(l.gamma_grads[i]) || isnan(gamma_grads[i])) {
			printf("Verification Failed: i = %zu, (cpu)%f != (gpu)%f\n", i, l.gamma_grads[i], gamma_grads[i]);
			wait_for_key_then_exit();
		}
	}
	printf("Gamma grads verification success.\n");
}