#include "hip/hip_runtime.h"
#include <stdio.h>
#include "xcuda.h"
#include "xallocs.h"
#include "utils.h"
#include "gemm.h"
#include <math.h>
#include "blas.h"


#ifdef __INTELLISENSE__
#define KARGS(...)
#define __syncthreads()
#define __shfl_down_sync(...) ( __VA_ARGS__ )
#else
#define KARGS(...) <<< __VA_ARGS__ >>>
#endif


void print_test_matrix(size_t rows, size_t cols, size_t channels, float* matrix);


#define TILE_SIZE 16


__global__ void gemm_kernel(
	const float* __restrict__ A,
	const float* __restrict__ B,
	float* __restrict__ C,
	int M, int N, int K,
	int A_offset, int B_offset, int C_offset)
{
	// Shared memory for A and B tiles
	__shared__ float A_shared[TILE_SIZE][TILE_SIZE];
	__shared__ float B_shared[TILE_SIZE][TILE_SIZE];

	// Row and column indices of the C element to work on
	int row = blockIdx.y * TILE_SIZE + threadIdx.y;
	int col = blockIdx.x * TILE_SIZE + threadIdx.x;

	float c_partial = 0.0F;

	// Loop over tiles of A and B to accumulate the result
	for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
		// Load tiles into shared memory
		int A_row = row;
		int A_col = t * TILE_SIZE + threadIdx.x;

		int B_row = t * TILE_SIZE + threadIdx.y;
		int B_col = col;

		A_shared[threadIdx.y][threadIdx.x] = (A_row < M && A_col < K) ? A[A_offset + A_row * K + A_col] : 0.0F;
		B_shared[threadIdx.y][threadIdx.x] = (B_row < K && B_col < N) ? B[B_offset + B_row * N + B_col] : 0.0F;

		__syncthreads();

		// Multiply the two tiles
		for (int i = 0; i < TILE_SIZE; ++i) {
			c_partial += A_shared[threadIdx.y][i] * B_shared[i][threadIdx.x];
		}

		__syncthreads();
	}

	// Write the result back to global memory
	if (row < M && col < N) {
		C[C_offset + row * N + col] += c_partial;
	}
}

void gemm_gpu(size_t M, size_t N, size_t K, float* A, float* B, float* C, int n_groups) {
	/*
	M = # of filters
	N = # of outputs per filter
	K = # of weights per filter (as if n_groups = 1)
	A = weight matrix (M * K)
	B = expanded input matrix (K * N)
	C = output dot products (M * N)
	*/
	if (n_groups > 1) {
		M = M / n_groups;  // # of filters per group
		K = K / n_groups;  // # of weights per filter per group
		dim3 threads(TILE_SIZE, TILE_SIZE);
		dim3 blocks((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
		for (int g = 0; g < n_groups; g++) {
			int a_offset = g * M * K;
			int b_offset = g * N * K;
			int c_offset = g * M * N;
			gemm_kernel KARGS(blocks, threads) (A, B, C, M, N, K, a_offset, b_offset, c_offset);
		}
	}
	else {
		dim3 threads(TILE_SIZE, TILE_SIZE);
		dim3 blocks((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
		gemm_kernel KARGS(blocks, threads) (A, B, C, M, N, K, 0, 0, 0);
	}
	
	CHECK_CUDA(hipPeekAtLastError());
}

void cuda_test_gemm(void) {
	/*
	M = # of filters
	N = # of outputs per filter
	K = # of weights per filter (if n_groups = 1)
	A = weight matrix (M * K)
	B = expanded input matrix (K * N)
	C = output dot products (M * N)
	*/
	int width = 320;
	int height = width;
	int channels = 16;
	if (width % 32 != 0) {
		printf("Input width must be a multiple of 32.\n");
		exit(EXIT_FAILURE);
	}

	size_t n_filters = 32;
	size_t pad = 1;
	size_t stride = 1;
	size_t ksize = 3;
	size_t out_size = (width + 2 * pad - ksize) / stride + 1; // square image

	size_t M = n_filters;
	size_t N = out_size * out_size;
	size_t K = ksize * ksize * channels;
	size_t n_groups = 2;

	if (M % n_groups > 0 || K % n_groups > 0) {
		printf("Cannot divide filters or weights evenly between groups.\n");
		(void)getchar();
		exit(EXIT_FAILURE);
	}

	float* A = (float*)xmalloc((size_t)(M * K / n_groups) * sizeof(float));
	float* B = (float*)xmalloc((size_t)(N * K) * sizeof(float));
	float* C = (float*)xcalloc((size_t)(M * N), sizeof(float));
	fill_array_rand_float(A, M * K / n_groups, 0., 1.);
	fill_array_rand_float(B, N * K, 0., 1.);

	float* d_a = 0;
	float* d_b = 0;
	float* d_c = 0;

	CHECK_CUDA(hipMalloc(&d_a, M * K / n_groups * sizeof(float)));
	CHECK_CUDA(hipMalloc(&d_b, N * K * sizeof(float)));
	CHECK_CUDA(hipMalloc(&d_c, M * N * sizeof(float)));

	CHECK_CUDA(hipMemcpy(d_a, A, M * K / n_groups * sizeof(float), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(d_b, B, N * K * sizeof(float), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemset(d_c, 0, M * N * sizeof(float)));


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	gemm_gpu(M, N, K, d_a, d_b, d_c, n_groups);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("gemm kernel execution time: %f ms\n", milliseconds);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	CHECK_CUDA(hipGetLastError());
	CHECK_CUDA(hipDeviceSynchronize());

	CHECK_CUDA(hipMemcpy(C, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_a));
	CHECK_CUDA(hipFree(d_b));
	CHECK_CUDA(hipFree(d_c));

	float* gemm_cpu = (float*)xcalloc(M * N, sizeof(float));
	gemm_groups(M, N, K, A, B, gemm_cpu, n_groups);
	free(A);
	free(B);
	float epsilon = 1e-5f;
	size_t zero_count = 0;
	printf("Verifiying......\n");
	for (size_t i = 0; i < M * N; i++) {
		//printf("%f : %f\n", gemm_cpu[i], C[i]);
		if (fabs(gemm_cpu[i] - C[i]) > epsilon || isnan(gemm_cpu[i]) || isnan(C[i])) {
			printf("Verification Failed: i = %zu, (gemm_cpu)%f != (gemm_gpu)%f\n", i, gemm_cpu[i], C[i]);
			wait_for_key_then_exit();
		}
		if (gemm_cpu[i] == 0.0F && C[i] == 0.0F) {
			zero_count++;
			printf("zero count: %zu\r", zero_count);
		}
	}
	printf("zero count: %zu\n", zero_count);
	printf("Verifiction Success!!!\n\n");
}

/*A[M*K], B[N*K], BT[K*N], C[M*N]*/
__global__ void gemm_atb_kernel(
	const float* __restrict__ A,
	const float* __restrict__ B,
	float* __restrict__ C,
	int M, int N, int K,
	int A_offset, int B_offset, int C_offset)
{
	// Shared memory for A and B tiles
	__shared__ float A_shared[TILE_SIZE][TILE_SIZE];
	__shared__ float B_shared[TILE_SIZE][TILE_SIZE];

	// Row and column indices of the C element to work on
	int row = blockIdx.y * TILE_SIZE + threadIdx.y;
	int col = blockIdx.x * TILE_SIZE + threadIdx.x;

	float c_partial = 0.0F;

	// Loop over tiles of A and B to accumulate the result
	for (int t = 0; t < (K + TILE_SIZE - 1) / TILE_SIZE; ++t) {
		// Load tile of A into shared memory
		int A_row = row;
		int A_col = t * TILE_SIZE + threadIdx.x;
		A_shared[threadIdx.y][threadIdx.x] = (A_row < M && A_col < K) ? A[A_offset + A_row * K + A_col] : 0.0F;

		// Load tile of B into shared memory with transposition
		// For transposition, we swap row and column when accessing global memory
		int B_col = t * TILE_SIZE + threadIdx.y; // Transposed row becomes column
		int B_row = col;                         // Transposed column becomes row
		B_shared[threadIdx.y][threadIdx.x] = (B_row < N && B_col < K) ? B[B_offset + B_row * K + B_col] : 0.0F;

		__syncthreads();

		// Multiply the two tiles
		for (int i = 0; i < TILE_SIZE; ++i) {
			c_partial += A_shared[threadIdx.y][i] * B_shared[i][threadIdx.x];
		}

		__syncthreads();
	}

	// Write the result back to global memory
	if (row < M && col < N) {
		C[C_offset + row * N + col] += c_partial;
	}
}

void gemm_atb_gpu(size_t M, size_t N, size_t K, float* A, float* B, float* C, int n_groups) {
	// M = # of filters
	// N = # of weights per filter (as if n_groups = 1)
	// K = # of outputs per filter
	// A = M * K (dC/dz grads)
	// B = N * K -> transpose -> K * N
	// C = M * N
	if (n_groups > 1) {
		M = M / n_groups; // # of filters per group
		N = N / n_groups; // # of weights per filter per group
		dim3 threads(TILE_SIZE, TILE_SIZE);
		dim3 blocks((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
		for (int g = 0; g < n_groups; g++) {
			int a_offset = g * M * K;
			int b_offset = g * N * K;
			int c_offset = g * M * N;
			gemm_atb_kernel KARGS(blocks, threads) (A, B, C, M, N, K, a_offset, b_offset, c_offset);
		}
	}
	else {
		dim3 threads(TILE_SIZE, TILE_SIZE);
		dim3 blocks((N + TILE_SIZE - 1) / TILE_SIZE, (M + TILE_SIZE - 1) / TILE_SIZE);
		gemm_atb_kernel KARGS(blocks, threads) (A, B, C, M, N, K, 0, 0, 0);
	}

	CHECK_CUDA(hipPeekAtLastError());
}

void cuda_test_gemm_atb(void) {
	int width = 320;
	int height = width;
	int channels = 16;
	if (width % 32 != 0) {
		printf("Input width must be a multiple of 32.\n");
		exit(EXIT_FAILURE);
	}

	size_t n_groups = 2;
	size_t n_filters = 32; // must be an even number
	size_t pad = 1;
	size_t stride = 1;
	size_t ksize = 3;
	size_t out_size = (width + 2 * pad - ksize) / stride + 1; // square image

	// M = # of filters
	// N = # of weights per filter (as if n_groups = 1) (ksize * ksize * input_channels)
	// K = out_w * out_h
	// A = M * K (dC/dz grads)
	// B = N * K -> transpose -> K * N (im2col?)
	// C = M * N (weight grads?)
	size_t M = n_filters;
	size_t N = ksize * ksize * channels;
	size_t K = out_size * out_size;
	
	if (M % n_groups > 0 || N % n_groups > 0) {
		printf("Cannot divide filters or weights evenly between groups.\n");
		(void)getchar();
		exit(EXIT_FAILURE);
	}
	float* A = (float*)xmalloc(M * K * sizeof(float));
	float* B = (float*)xmalloc(N * K * sizeof(float));
	float* C = (float*)xcalloc(M * (N / n_groups), sizeof(float));

	fill_array_rand_float(A, M * K, 0., 1.);
	fill_array_rand_float(B, N * K, 0., 1.);

	float* d_a = 0;
	float* d_b = 0;
	float* d_c = 0;

	CHECK_CUDA(hipMalloc(&d_a, M * K * sizeof(float)));
	CHECK_CUDA(hipMalloc(&d_b, N * K * sizeof(float)));
	CHECK_CUDA(hipMalloc(&d_c, M * (N / n_groups) * sizeof(float)));

	CHECK_CUDA(hipMemcpy(d_a, A, M * K * sizeof(float), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(d_b, B, N * K * sizeof(float), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemset(d_c, 0, M * (N / n_groups) * sizeof(float)));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	gemm_atb_gpu(M, N, K, d_a, d_b, d_c, n_groups);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("gemm_atb kernel execution time: %f ms\n", milliseconds);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	CHECK_CUDA(hipGetLastError());

	CHECK_CUDA(hipDeviceSynchronize());

	CHECK_CUDA(hipMemcpy(C, d_c, sizeof(float) * M * (N / n_groups), hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_a));
	CHECK_CUDA(hipFree(d_b));
	CHECK_CUDA(hipFree(d_c));


	float* gemm_atb_cpu = (float*)xcalloc(M * (N / n_groups), sizeof(float));
	gemm_atb_groups(M, N, K, A, B, gemm_atb_cpu, n_groups);
	free(A);
	free(B);
	float epsilon = 1e-5f;
	size_t zero_count = 0;
	printf("Verifiying......\n");
	for (size_t i = 0; i < M * (N / n_groups); i++) {
		//printf("%f : %f\n", gemm_atb_cpu[i], C[i]);
		if (fabs(gemm_atb_cpu[i] - C[i]) > epsilon || isnan(gemm_atb_cpu[i]) || isnan(C[i])) {
			printf("\nVerification Failed: i = %zu, (gemm_atb_cpu)%f != (gemm_atb_gpu)%f\n", i, gemm_atb_cpu[i], C[i]);
			wait_for_key_then_exit();
		}
		if (gemm_atb_cpu[i] == 0.0F && C[i] == 0.0F) {
			zero_count++;
			printf("zero count: %zu\r", zero_count);
		}
	}
	printf("zero count: %zu\n", zero_count);
	printf("Verifiction Success!!!\n\n");
}

__global__ void gemm_tab_kernel(
	const float* __restrict__ A,
	const float* __restrict__ B,
	float* __restrict__ C,
	int M, int N, int K,
	int A_offset, int B_offset, int C_offset)
{
	__shared__ float A_shared[TILE_SIZE][TILE_SIZE];
	__shared__ float B_shared[TILE_SIZE][TILE_SIZE];

	// These should map to the output C dimensions:
	int row = blockIdx.y * TILE_SIZE + threadIdx.y;  // N rows
	int col = blockIdx.x * TILE_SIZE + threadIdx.x;  // K columns

	float c_partial = 0.0F;

	for (int t = 0; t < (M + TILE_SIZE - 1) / TILE_SIZE; ++t) {
		// For A (transposed):
		int A_col = t * TILE_SIZE + threadIdx.x;  // M columns
		int A_row = row;                          // N rows
		A_shared[threadIdx.y][threadIdx.x] = (A_row < N && A_col < M) ? A[A_offset + A_col * N + A_row] : 0.0F;

		// For B:
		int B_row = t * TILE_SIZE + threadIdx.y;  // M rows
		int B_col = col;                          // K columns
		B_shared[threadIdx.y][threadIdx.x] = (B_row < M && B_col < K) ? B[B_offset + B_row * K + B_col] : 0.0F;

		__syncthreads();

		for (int i = 0; i < TILE_SIZE; ++i) {
			c_partial += A_shared[threadIdx.y][i] * B_shared[i][threadIdx.x];
		}

		__syncthreads();
	}

	if (row < N && col < K) {
		C[C_offset + row * K + col] += c_partial;
	}
}

void gemm_tab_gpu(size_t M, size_t N, size_t K, float* A, float* B, float* C, int n_groups) {
	// M = # of filters
	// N = # of weights per filter (as if n_groups = 1)
	// K = # of outputs per filter
	// A = M * N -> transpose -> N * M (weights)
	// B = M * K (dC/dz grads)
	// C = N * K (col'd array to go through col2im)
	if (n_groups > 1) {
		M = M / n_groups; // # of filters per group
		N = N / n_groups; // # of weights per filter per group
		dim3 threads(TILE_SIZE, TILE_SIZE);
		dim3 blocks((K + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);
		for (int g = 0; g < n_groups; g++) {
			int a_offset = g * M * N;
			int b_offset = g * M * K;
			int c_offset = g * N * K;
			gemm_tab_kernel KARGS(blocks, threads) (A, B, C, M, N, K, a_offset, b_offset, c_offset);
		}
	}
	else {
		dim3 threads(TILE_SIZE, TILE_SIZE);
		dim3 blocks((K + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);
		gemm_tab_kernel KARGS(blocks, threads) (A, B, C, M, N, K, 0, 0, 0);
	}

	CHECK_CUDA(hipPeekAtLastError());
}

void cuda_test_gemm_tab(void) {
	int width = 320;
	int height = width;
	int channels = 16;
	if (width % 32 != 0) {
		printf("Input width must be a multiple of 32.\n");
		exit(EXIT_FAILURE);
	}

	size_t n_groups = 2;
	size_t n_filters = 32; // must be an even number
	size_t pad = 1;
	size_t stride = 1;
	size_t ksize = 3;
	size_t out_size = (width + 2 * pad - ksize) / stride + 1; // square image

	// M = # of filters
	// N = # of weights per filter (as if n_groups = 1)
	// K = out_w * out_h
	// A = M * N -> transpose -> N * M (weights)
	// B = M * K (dC/dz grads)
	// C = N * K (col'd array to go through col2im)
	size_t M = n_filters;
	size_t N = ksize * ksize * channels;
	size_t K = out_size * out_size;
	
	if (M % n_groups > 0 || N % n_groups > 0) {
		printf("Cannot divide filters or weights evenly between groups.\n");
		(void)getchar();
		exit(EXIT_FAILURE);
	}
	float* A = (float*)xmalloc(M * (N / n_groups) * sizeof(float));
	float* B = (float*)xmalloc(M * K * sizeof(float));
	float* C = (float*)xcalloc(N * K, sizeof(float));
	
	fill_array_rand_float(A, M * (N / n_groups), 0., 1.);
	fill_array_rand_float(B, M * K, 0., 1.);

	float* d_a = 0;
	float* d_b = 0;
	float* d_c = 0;

	CHECK_CUDA(hipMalloc(&d_a, M * (N / n_groups) * sizeof(float)));
	CHECK_CUDA(hipMalloc(&d_b, M * K * sizeof(float)));
	CHECK_CUDA(hipMalloc(&d_c, N * K * sizeof(float)));

	CHECK_CUDA(hipMemcpy(d_a, A, M * (N / n_groups) * sizeof(float), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemcpy(d_b, B, M * K * sizeof(float), hipMemcpyHostToDevice));
	CHECK_CUDA(hipMemset(d_c, 0, N * K * sizeof(float)));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	gemm_tab_gpu(M, N, K, d_a, d_b, d_c, n_groups);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("gemm_tab kernel execution time: %f ms\n", milliseconds);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	CHECK_CUDA(hipGetLastError());

	CHECK_CUDA(hipDeviceSynchronize());

	CHECK_CUDA(hipMemcpy(C, d_c, sizeof(float) * N * K, hipMemcpyDeviceToHost));
	CHECK_CUDA(hipFree(d_a));
	CHECK_CUDA(hipFree(d_b));
	CHECK_CUDA(hipFree(d_c));

	float* gemm_tab_cpu = (float*)xcalloc(N * K, sizeof(float));
	gemm_tab_groups(M, N, K, A, B, gemm_tab_cpu, n_groups);
	free(A);
	free(B);
	float epsilon = 1e-5f;
	printf("Verifiying......\n");
	size_t zero_count = 0;
	for (size_t i = 0; i < N * K; i++) {
		if (fabs(gemm_tab_cpu[i] - C[i]) > epsilon || isnan(gemm_tab_cpu[i]) || isnan(C[i])) {
			printf("Verification Failed: i = %zu, (gemm_tab_cpu)%f != (gemm_tab_gpu)%f\n", i, gemm_tab_cpu[i], C[i]);
			wait_for_key_then_exit();
		}
		if (gemm_tab_cpu[i] == 0.0F && C[i] == 0.0F) {
			zero_count++;
			printf("zero count: %zu\r", zero_count);
		}
	}
	printf("zero count: %zu\n", zero_count);
	printf("Verifiction Success!!!\n\n");
}

void cuda_test_all_gemms(void) {
	cuda_test_gemm();
	cuda_test_gemm_atb();
	cuda_test_gemm_tab();
}



void print_test_matrix(size_t rows, size_t cols, size_t channels, float* matrix) {
	for (size_t ch = 0; ch < channels; ch++) {
		printf("Channel: %zu\n", ch);
		for (size_t r = 0; r < rows; r++) {
			printf("%0.1f", matrix[ch * cols * rows + r * cols]);
			for (size_t c = 1; c < cols; c++) {
				printf(", %0.1f", matrix[ch * cols * rows + r * cols + c]);
			}
			printf("\n");
		}
		printf("\n");
	}
}