#include "hip/hip_runtime.h"
#include <stdio.h>
#include "xcuda.h"
#include "xallocs.h"
#include "utils.h"
#include "im2col.h"

#ifdef __INTELLISENSE__
#define KARGS(...)
#define cuda_syncthreads()
#else
#define KARGS(...) <<< __VA_ARGS__ >>>
#define cuda_syncthreads() __syncthreads()
#endif


__global__ void im2col_kernel(const float* data_im, const int height, const int width, const int channels,
    const int ksize, const int pad, const int stride,
    const int height_out, const int width_out, float* data_col,
    int n) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < n) {
        const int col_out = index % width_out;
        const int row_out = (index / width_out) % height_out;
        const int ch_in = (index / width_out / height_out) % channels;

        // Calculate corresponding top-left position in input
        const int row_in = row_out * stride - pad;
        const int col_in = col_out * stride - pad;

        const int data_im_offset = (ch_in * height + row_in) * width + col_in;

        const int data_col_offset = ((ch_in * ksize * ksize) * height_out * width_out) + (row_out * width_out + col_out);

        for (int krow = 0; krow < ksize; ++krow) {
            for (int kcol = 0; kcol < ksize; ++kcol) {
                const int row = row_in + krow;
                const int col = col_in + kcol;

                const int data_col_index = data_col_offset + ((krow * ksize + kcol) * height_out * width_out);

                if (row >= 0 && row < height && col >= 0 && col < width) {
                    data_col[data_col_index] = data_im[data_im_offset + krow * width + kcol];
                }
                else {
                    data_col[data_col_index] = 0.0F;
                }
            }
        }
    }
}

// Pad has to equal (ksize - 1) / 2. Stride must = 1 (for now). Block size must = input width.
__global__ void im2col_kernel_shared(const float* data_im, const int height, const int width, const int channels,
    const int ksize, const int pad, const int stride,
    const int height_col, const int width_col, float* data_col,
    int n) {

    extern __shared__ float shared_input[];  // Equals width_col * sizeof(float)

    int tx = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tx;

    // assert n == height_col * width_col * channels * ksize;

    // Each thread is responsible for copying one row of the kernel window
    if (index < n) {

        const int col_out = index % width_col;
        const int row_out = (index / width_col) % height_col;
        const int krow = (index / width_col / height_col) % ksize;
        const int ch_in = (index / width_col / height_col / ksize) % channels;

        const int row_in = row_out * stride - pad + krow;

        const int base_data_col_index = (((ch_in * ksize + krow) * ksize) * height_col + row_out) * width_col + col_out;

        shared_input[tx] = data_im[(ch_in * height + row_in) * width + tx];

        cuda_syncthreads();

        for (int kcol = 0; kcol < ksize; ++kcol) {
            int col_in = col_out * stride - pad + kcol;
            int data_col_index = base_data_col_index + kcol * height_col * width_col;

            if (row_in >= 0 && row_in < height && col_in >= 0 && col_in < width) {
                data_col[data_col_index] = shared_input[col_in];
            }
            else {
                data_col[data_col_index] = 0.0F;
            }
        }
    }
}

void cuda_test_im2col(void) {
    int width = 320;
    int height = 320;
    int channels = 80;
    if (width % 32 != 0) {
        printf("Input width must be a multiple of 32.\n");
        wait_for_key_then_exit();
    }
    int im_n = width * height * channels;
    float* im = (float*)calloc(im_n, sizeof(float));
    if (!im) {
        fprintf(stderr, "Failed to calloc im.");
        exit(EXIT_FAILURE);
    }
    int pad = 1;
    int stride = 1;
    int ksize = 3;
    int out_w = (width + pad * 2 - ksize) / stride + 1;
    int out_h = (height + pad * 2 - ksize) / stride + 1;

    int dst_w = out_w * out_h;
    int dst_h = ksize * ksize * channels;
    int dst_n = dst_w * dst_h;
    float* col = (float*)calloc(dst_n, sizeof(float));
    if (!col) {
        fprintf(stderr, "Failed to calloc col.");
        exit(EXIT_FAILURE);
    }

    fill_array_rand_float(im, im_n, 0, 1);

    float* d_im = 0;
    float* d_col = 0;

    CHECK_CUDA(hipMalloc(&d_im, im_n * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_col, dst_n * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_im, im, sizeof(float) * im_n, hipMemcpyHostToDevice));

    int threads_per_block = 512;  // Required for the shared mem im2col kernel to work
    int num_cuda_kernels = dst_n;
    int num_blocks = (num_cuda_kernels + threads_per_block - 1) / threads_per_block;

    printf("num_blocks = %d\n", num_blocks);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    im2col_kernel KARGS(num_blocks, threads_per_block) (d_im, height, width, channels,
        ksize, pad, stride,
        out_h, out_w, d_col,
        num_cuda_kernels);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel execution time (no shared mem): %f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(col, d_col, sizeof(float) * dst_n, hipMemcpyDeviceToHost));

    //pprint_mat(col, dst_w, dst_h, 1);
    float* col_cpu = (float*)calloc(dst_n, sizeof(float));
    if (!col_cpu) {
        fprintf(stderr, "Failed to calloc col_cpu.");
        exit(EXIT_FAILURE);
    }
    im2col(im, channels, height, width, ksize, pad, stride, col_cpu);

    printf("Verifiying......\n");
    for (size_t i = 0; i < dst_n; i++) {
        //printf("%f =? %f\n", col_cpu[i], col[i]);
        if (col_cpu[i] != col[i]) {
            printf("Verification Failed: i = %d, (col_cpu)%f != (col_gpu)%f\n", i, col_cpu[i], col[i]);
            wait_for_key_then_exit();
        }
    }
    printf("Verifiction Success!!!\n");
}

void cuda_test_im2col_shared(void) {
    int width = 320;
    int height = 320;
    int channels = 80;
    if (width % 32 != 0) {
        printf("Input width must be a multiple of 32.\n");
        wait_for_key_then_exit();
    }
    int im_n = width * height * channels;
    float* im = (float*)calloc(im_n, sizeof(float));
    if (!im) {
        fprintf(stderr, "Failed to calloc im.");
        exit(EXIT_FAILURE);
    }
    int pad = 1;
    int stride = 1;
    int ksize = 3;
    int out_w = (width + pad * 2 - ksize) / stride + 1;
    int out_h = (height + pad * 2 - ksize) / stride + 1;
    if (width != out_w) {
        printf("Input width must be equal to output width. (%d != %d)", width, out_w);
        return;
    }
    int dst_w = out_w * out_h;
    int dst_h = ksize * ksize * channels;
    int dst_n = dst_w * dst_h;
    float* col = (float*)calloc(dst_n, sizeof(float));
    if (!col) {
        fprintf(stderr, "Failed to calloc col.");
        exit(EXIT_FAILURE);
    }

    fill_array_rand_float(im, im_n, 0, 1);

    float* d_im = 0;
    float* d_col = 0;

    CHECK_CUDA(hipMalloc(&d_im, im_n * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_col, dst_n * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_im, im, sizeof(float) * im_n, hipMemcpyHostToDevice));

    int threads_per_block = out_w;  // Required for the shared mem im2col kernel to work
    int num_cuda_kernels = out_h * out_w * channels * ksize;
    int num_blocks = (num_cuda_kernels + threads_per_block - 1) / threads_per_block;

    size_t shared_mem_size = out_w * sizeof(float);
    printf("num_blocks = %d, width = %d, shared_mem_size = %zu\n", num_blocks, width, shared_mem_size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    im2col_kernel_shared KARGS(num_blocks, threads_per_block, shared_mem_size) (d_im, height, width, channels,
        ksize, pad, stride,
        out_h, out_w, d_col,
        num_cuda_kernels);


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel execution time (shared mem kernel): %f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(col, d_col, sizeof(float) * dst_n, hipMemcpyDeviceToHost));
    
    //pprint_mat(col, dst_w, dst_h, 1);
    float* col_cpu = (float*)calloc(dst_n, sizeof(float));
    if (!col_cpu) {
        fprintf(stderr, "Failed to calloc col_cpu.");
        exit(EXIT_FAILURE);
    }
    im2col(im, channels, height, width, ksize, pad, stride, col_cpu);

    printf("Verifiying......\n");
    for (size_t i = 0; i < dst_n; i++) {
        //printf("%f =? %f\n", col_cpu[i], col[i]);
        if (col_cpu[i] != col[i]) {
            printf("Verification Failed: i = %d, (col_cpu)%f != (col_gpu)%f\n", i, col_cpu[i], col[i]);
            wait_for_key_then_exit();
        }
    }
    printf("Verifiction Success!!!\n");
}