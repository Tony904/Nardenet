#include "hip/hip_runtime.h"
#include <stdio.h>
#include "xcuda.h"
#include "xallocs.h"
#include "utils.h"
#include "im2col.h"
#include <math.h>


#ifdef __INTELLISENSE__
#define KARGS(...)
#define __syncthreads()
#else
#define KARGS(...) <<< __VA_ARGS__ >>>
#endif

#ifndef min
#define min(x, y) ((x > y) ? y : x)
#endif


// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE
__global__ void col2im_kernel(const float* __restrict__ data_col,
    const int width_col, const int height_col,
    const int ksize, const int pad, const int stride,
    const int width, const int height,
    float* __restrict__ data_im,
    const int n) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for (; index < n; index += blockDim.x * gridDim.x) {
        float val = 0;
        int w = index % width + pad;
        int h = (index / width) % height + pad;
        int c = index / (width * height);
        int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
        int w_col_end = min(w / stride + 1, width_col);
        int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
        int h_col_end = min(h / stride + 1, height_col);
        int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
        int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
        int coeff_w_col = (1 - stride * height_col * width_col);
        for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
            for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
            }
        }
        data_im[index] += val;
    }
}

void col2im_gpu(float* data_col, int channels, int ksize, int pad, int stride, int width, int height, float* data_im, int n) {
    int out_size = (width + 2 * pad - ksize) / stride + 1;
    int grid_size = GET_GRIDSIZE(n, BLOCKSIZE);
    col2im_kernel KARGS(grid_size, BLOCKSIZE) (data_col, out_size, out_size,
        ksize, pad, stride,
        height, width,
        data_im,
        n);
    CHECK_CUDA(hipPeekAtLastError());
}

void cuda_test_col2im(void) {
    int width = 64;
    int height = width;
    int channels = 16;
    if (width % 32 != 0) {
        printf("Input width must be a multiple of 32.\n");
        exit(EXIT_FAILURE);
    }

    size_t pad = 1;
    size_t stride = 1;
    size_t ksize = 3;
    size_t col_size = (width + 2 * pad - ksize) / stride + 1; // square image
    size_t col_n = ksize * ksize * channels * col_size * col_size;
    float* col = (float*)xmalloc(col_n * sizeof(float));
    fill_array_rand_float(col, col_n, 0., 1.);

    int im_n = width * height * channels;
    float* im = (float*)xmalloc(im_n * sizeof(float));

    float* d_im = 0;
    float* d_col = 0;

    CHECK_CUDA(hipMalloc(&d_im, im_n * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_col, col_n * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_col, col, sizeof(float) * col_n, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    col2im_gpu(d_col, channels, ksize, pad, stride, width, height, d_im, im_n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("col2im kernel execution time: %f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(im, d_im, sizeof(float) * im_n, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(d_col));
    CHECK_CUDA(hipFree(d_im));

    //pprint_mat(col, dst_w, dst_h, 1);
    float* im_cpu = (float*)xcalloc(im_n, sizeof(float));
    col2im(col, channels, height, width, ksize, pad, stride, im_cpu);

    float epsilon = 1e-5f;
    size_t zero_count = 0;
    printf("Verifiying......\n");
    for (size_t i = 0; i < im_n; i++) {
        //printf("%f =? %f\n", col_cpu[i], col[i]);
        if (fabs(im_cpu[i] - im[i]) > epsilon) {
            printf("Verification Failed: i = %zu, (im_cpu)%f != (im_gpu)%f\n", i, im_cpu[i], im[i]);
            wait_for_key_then_exit();
        }
        if (im_cpu[i] == 0.0F && im[i] == 0.0F) {
            zero_count++;
            printf("zero count: %zu\r", zero_count);
        }
    }
    printf("zero count: %zu\n", zero_count);
    printf("Verifiction Success!!!\n");
}

/*******************************************
                   IM2COL
*******************************************/

__global__ void im2col_kernel_no_share(const float* __restrict__ data_im,
    const int height, const int width, const int channels,
    const int ksize, const int pad, const int stride,
    const int height_out, const int width_out,
    float* __restrict__ data_col,
    int n) {
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < n; index += blockDim.x * gridDim.x) {
        const int col_out = index % width_out;
        const int row_out = (index / width_out) % height_out;
        const int ch_in = (index / width_out / height_out) % channels;
        const int row_in = row_out * stride - pad;
        const int col_in = col_out * stride - pad;
        const int data_im_offset = (ch_in * height + row_in) * width + col_in;
        const int data_col_offset = ((ch_in * ksize * ksize) * height_out * width_out) + (row_out * width_out + col_out);
        for (int krow = 0; krow < ksize; ++krow) {
            for (int kcol = 0; kcol < ksize; ++kcol) {
                const int row = row_in + krow;
                const int col = col_in + kcol;
                const int data_col_index = data_col_offset + ((krow * ksize + kcol) * height_out * width_out);
                if (row >= 0 && row < height && col >= 0 && col < width) {
                    data_col[data_col_index] = data_im[data_im_offset + krow * width + kcol];
                }
                else data_col[data_col_index] = 0.0F;
            }
        }
    }
}

__global__ void im2col_kernel(
    const float* __restrict__ input, float* __restrict__ output,
    const int channels, const int height, const int width,
    const int ksize, const int stride, const int pad,
    const int out_height, const int out_width) {

    extern __shared__ float shared_input[];

    int block_row = blockIdx.y * blockDim.y;
    int block_col = blockIdx.x * blockDim.x;
    int c = blockIdx.z;
    int thread_row = threadIdx.y;
    int thread_col = threadIdx.x;
    int start_input_row = block_row * stride - pad;
    int start_input_col = block_col * stride - pad;
    int shared_height = blockDim.y + ksize - 1;
    int shared_width = blockDim.x + ksize - 1;

    // Load data from global memory to shared memory
    // Each thread may need to load multiple elements to cover the padded region
    for (int i = thread_row; i < shared_height; i += blockDim.y) {
        for (int j = thread_col; j < shared_width; j += blockDim.x) {
            int input_row = start_input_row + i;
            int input_col = start_input_col + j;
            if (input_row >= 0 && input_row < height && input_col >= 0 && input_col < width) {
                shared_input[i * shared_width + j] = input[c * height * width + input_row * width + input_col];
            }
            else shared_input[i * shared_width + j] = 0.0F; // Zero padding
        }
    }
    __syncthreads();
    if (thread_row < out_height && thread_col < out_width) {
        int output_index = (block_row + thread_row) * out_width + (block_col + thread_col);
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                // Calculate the position in shared memory to read from
                int local_row = thread_row + i;
                int local_col = thread_col + j;
                int output_channel_offset = (c * ksize * ksize + i * ksize + j) * out_height * out_width;
                output[output_channel_offset + output_index] = shared_input[local_row * shared_width + local_col];
            }
        }
    }
}

void im2col_gpu(float* data_im, float* data_col, int channels, int h, int w, int ksize, int stride, int pad, int out_h, int out_w) {
    dim3 block_size(16, 16);
    dim3 grid_size((out_w + block_size.x - 1) / block_size.x, (out_h + block_size.y - 1) / block_size.y, channels);
    size_t shared_memory_size = (block_size.x + 2 * pad) * (block_size.y + 2 * pad) * sizeof(float);
    im2col_kernel KARGS(grid_size, block_size, shared_memory_size) (data_im, data_col, channels, h, w, ksize, stride, pad, out_h, out_w);
    CHECK_CUDA(hipPeekAtLastError());
}

void cuda_test_im2col(void) {
    int width = 64;
    int height = 64;
    int channels = 3;
    if (width % 32 != 0) {
        printf("Input width must be a multiple of 32.\n");
        exit(EXIT_FAILURE);
    }
    int im_n = width * height * channels;
    float* im = (float*)xmalloc(im_n * sizeof(float));
    int pad = 1;
    int stride = 1;
    int ksize = 3;
    if (pad != (ksize - 1) / 2) {
        printf("Pad must equal (ksize - 1) / 2.\n");
        exit(EXIT_FAILURE);
    }
    int out_w = (width + pad * 2 - ksize) / stride + 1;
    int out_h = (height + pad * 2 - ksize) / stride + 1;

    int dst_w = out_w * out_h;
    int dst_h = ksize * ksize * channels;
    int dst_n = dst_w * dst_h;
    float* col = (float*)xcalloc(dst_n, sizeof(float));

    fill_array_rand_float(im, im_n, 0., 1.);

    float* d_im = 0;
    float* d_col = 0;

    CHECK_CUDA(hipMalloc(&d_im, im_n * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_col, dst_n * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_im, im, sizeof(float) * im_n, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    im2col_gpu(d_im, d_col, channels, height, width, ksize, stride, pad, out_h, out_w);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Kernel execution time (shared mem kernel claude): %f ms\n", milliseconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(col, d_col, sizeof(float) * dst_n, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipFree(d_col));
    CHECK_CUDA(hipFree(d_im));

    float* col_cpu = (float*)xcalloc(dst_n, sizeof(float));
    im2col(im, channels, height, width, ksize, pad, stride, col_cpu);  // gives known correct result

    printf("Verifiying......\n");
    float epsilon = 1e-5f;
    size_t zero_count = 0;
    for (size_t i = 0; i < dst_n; i++) {
        printf("%f =? %f\n", col_cpu[i], col[i]);
        if (fabs(col_cpu[i] - col[i]) > epsilon || isnan(col_cpu[i]) || isnan(col[i])) {
            printf("Verification Failed: i = %zu, (col_cpu)%f != (col_gpu)%f\n", i, col_cpu[i], col[i]);
            wait_for_key_then_exit();
        }
        if (col_cpu[i] == 0.0F && col[i] == 0.0F) {
            zero_count++;
            printf("zero count: %zu\r", zero_count);
        }
    }
    printf("zero count: %zu\n", zero_count);
    printf("Verifiction Success!!!\n");
}